#include "hip/hip_runtime.h"
/*
 * Device code.
 */

#ifndef _GPUAO_KERNEL_H_
#define _GPUAO_KERNEL_H_
#include "cutil_inline.h"
#include "cutil_math.h"
#include "GPUAO_kernel.cuh"

#define BLOCKDIM 256
__constant__ AOParams params;
__global__ void
parpareMesh(float3* DPos,uint* DFaces, float3* DNormal,float* DArea)
{
	uint index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
	if (index<=params.TotalVerts)
	{
		float3 iNormal=make_float3(0.0f, 0.0f, 0.0f);
		float PArea=0.0f;
		uint numNormals = 0;
		for (uint f = 0; f < params.TotalFaces; f++) {
			for (uint fi = 0; fi < 3; fi++) {
				if (DFaces[f*3+fi]== index) {
					float3 V0=DPos[DFaces[f*3]];
					float3 V1=DPos[DFaces[f*3+1]];
					float3 V2=DPos[DFaces[f*3+2]];
					float3 fNormal=cross((V1-V0),(V2-V1)); 
					PArea+=length(fNormal)/2.0f;
					iNormal += fNormal;
					numNormals++;
				}
			}
		}
		iNormal = normalize(iNormal);
		
		if (numNormals>0)
		{
			DNormal[index] = iNormal ;
			DArea[index]=PArea/((float)numNormals*M_PI);
		}
		else
		{
			DNormal[index] = make_float3(0.0f, 0.0f, 0.0f);
			DArea[index]=0.0f;
		}

	}
}
__global__ void
AOProcess(float3* DPos, float3* DNormal,float* DArea,float* DColor,uint pass)
{
	int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
	if (index<=params.TotalVerts)
	{
		__shared__ float3 posTmp;
		__shared__ float3 NormalTmp;
		float  areaTmp;
		float3 v;
		float d2;
		float value;
		float total=0.0f; 
		
		posTmp=DPos[index];
		NormalTmp=DNormal[index];
		for(int i=0;i<params.TotalVerts;i++)
		{
			areaTmp=DArea[i];
			v=DPos[i]-posTmp;
			if (length(v)>params.Distance||i==index||length(v)==0)
				continue;
			d2=dot(v,v)+1e-16;
			if(d2<-4*areaTmp)
			{
				DArea[i]=0.0f;
			}
			v*= rsqrt(d2);
			value=(1.0f -rsqrt(abs(areaTmp)/d2 + 1.0f))*saturate(dot(DNormal[i], v))*saturate(3.0f*dot(NormalTmp, v));
			if (pass==2)
				value*=DColor[i];
			NormalTmp-= value*v;
			total += value;
		}
		DNormal[index]=NormalTmp;
		if (pass==1)
			DColor[index] = saturate(1.0f-total);
		else
			DColor[index] = DColor[index]*0.4f+ saturate(1.0f - total)*0.6f;
	}
}
extern "C"
{
	void cudaInit()
	{   
		hipSetDevice( cutGetMaxGflopsDeviceId() );
	}
	void allocateArray(void **devPtr, size_t size)
	{
		cutilSafeCall(hipMalloc(devPtr, size));
	}
	void freeArray(void *devPtr)
	{
		cutilSafeCall(hipFree(devPtr));
	}
	void threadSync()
	{
		cutilSafeCall(hipDeviceSynchronize());
	}
	void copyArrayFromDevice(void* host, const void* device, int size)
	{   
		cutilSafeCall(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
	}
	void copyArrayToDevice(void* device, const void* host, int offset, int size)
	{
		cutilSafeCall(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
	}
	void setParameters(AOParams *hostParams)
	{
		// copy parameters to constant memory
		cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(AOParams)) );
	}

	int iDivUp(int a, int b){
		return (a % b != 0) ? (a / b + 1) : (a / b);
	}
	// compute grid and thread block size for a given number of elements
	void computeGridSize(int n, int blockSize, int &numBlocks, int &numThreads)
	{
		numThreads = min(blockSize, n);
		numBlocks = iDivUp(n, numThreads);
	}
	void prepareMeshSystem(float* dPos,uint* dFaces, float* dNor, float* dArea,uint numVerts)
	{
		int numThreads, numBlocks;
		computeGridSize(numVerts, BLOCKDIM , numBlocks, numThreads);
		// execute the kernel
		
		//first prepare the mesh for normal and area
		parpareMesh<<< numBlocks, numThreads >>>((float3*) dPos, dFaces,(float3*) dNor,dArea);
		cutilCheckMsg("Kernel execution failed");
		
	}
	void integrateSystem(float* dPos, float* dNor, float* dArea, float* dColor,uint numVerts,uint totalPass)
	{
		int numThreads, numBlocks;
		computeGridSize(numVerts, BLOCKDIM, numBlocks, numThreads);
		// execute the kernel
		
		//process the AO with different pass
		for(uint pass=1;pass<=totalPass;pass++)
		{
			AOProcess<<< numBlocks, numThreads >>>((float3*) dPos,(float3*) dNor,dArea, dColor,pass);
			// check if kernel invocation generated an error
			cutilCheckMsg("Kernel execution failed");
		}
	}
}
#endif